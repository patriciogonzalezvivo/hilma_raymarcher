#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <chrono>

#include <hip/hip_runtime.h>
#include <>
// #include "hiprand/hiprand_kernel.h"
#define BLOCK_SIZE 8

#include "lygia/color/palette/heatmap.cuh"
#include "lygia/math.cuh"

#include "hilma/types/Image.h"
#include "hilma/io/png.h"

__global__ void render(int _width, int _height, float *_pixels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= _width || y >= _height)
        return;

    float4 color = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
    float2 pixel = make_float2(1.0 / _width, 1.0 / _height);
    float2 st = make_float2(x, y) * pixel;

    color.x = st.x;
    color.y = st.y;

    float3 hue = heatmap(st.x);
    color.x = hue.x;
    color.y = hue.y;
    color.z = hue.z;

    // return color
    _pixels[x * 4 + 4 * y * _width + 0] = color.x;
    _pixels[x * 4 + 4 * y * _width + 1] = color.y;
    _pixels[x * 4 + 4 * y * _width + 2] = color.z;
    _pixels[x * 4 + 4 * y * _width + 3] = color.w;
}

int main(int argc, char **argv) {
    int width = 1920;
    int height = 1080;

    // Create a buffer of pixels to be process on __device__
    float *d_pixels;
    hipMalloc(&d_pixels, 4 * width * height * sizeof(float));

    // Hand the pixels to a kernel to be process on __device__ 
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(width / threads.x + 1, height / threads.y + 1);
    render<<<blocks, threads>>>(width, height, d_pixels);

    // Copy pixels processed on __device__ into a HILMA Image on __host__
    hilma::Image image = hilma::Image(width, height, 4);
    hipMemcpy(&image[0], d_pixels, 4 * width * height * sizeof(float), hipMemcpyDeviceToHost);

    // Save image
    hilma::savePng("image.png", image);

    // Free pixels on __device__
    hipFree(d_pixels);

    return 0;
}