#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <chrono>

#include <hip/hip_runtime.h>
#include <>
// #include "hiprand/hiprand_kernel.h"
#include "cutil_math.h"
#define BLOCK_SIZE 8

#include "hilma/types/Image.h"
#include "hilma/io/png.h"

__global__ void render(int _width, int _height, float *_pixels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= _width || y >= _height)
        return;

    float4 color = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
    float2 pixel = make_float2(1.0 / _width, 1.0 / _height);
    float2 st = make_float2(x, y);
    st = st * pixel;

    color.x = st.x;
    color.y = st.y;

    // return color
    _pixels[x * 4 + 4 * y * _width + 0] = color.x;
    _pixels[x * 4 + 4 * y * _width + 1] = color.y;
    _pixels[x * 4 + 4 * y * _width + 2] = color.z;
    _pixels[x * 4 + 4 * y * _width + 3] = color.w;
}

int main(int argc, char **argv)
{
    int width = 1920;
    int height = 1080;

    hilma::Image image = hilma::Image(width, height, 4);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(width / threads.x + 1, height / threads.y + 1);

    float *device_pixels;
    hipMalloc(&device_pixels, 4 * width * height * sizeof(float));

    render<<<blocks, threads>>>(width, height, device_pixels);

    float *host_pixels = (float*)malloc(4 * width * height * sizeof(float));
    hipMemcpy(host_pixels, device_pixels, 4 * width * height * sizeof(float), hipMemcpyDeviceToHost);

    image.setColors(host_pixels, width * height, 4);
    hilma::savePng("image.png", image);

    hipFree(device_pixels);
    free(host_pixels);

    return 0;
}