#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <chrono>

#include <hip/hip_runtime.h>
#include <>
// #include "hiprand/hiprand_kernel.h"
#define BLOCK_SIZE 8

#include "hilma/types/Image.h"
#include "hilma/ops/convert_image.h"
#include "hilma/io/png.h"

#include "lygia/math/sum.cuh"
#include "lygia/math/cross.cuh"
#include "lygia/math/floor.cuh"
#include "lygia/math/fract.cuh"
#include "lygia/math/reflect.cuh"
#include "lygia/math/normalize.cuh"
#include "lygia/math/operations.cuh"
#include "lygia/math/smoothstep.cuh"
#include "lygia/space/ratio.cuh"
#include "lygia/sdf/sphereSDF.cuh"
#include "lygia/sdf/planeSDF.cuh"
#include "lygia/sdf/opUnion.cuh"
#include "lygia/color/palette/hue.cuh"

#define RAYMARCH_MAP_TYPE float4
#define RAYMARCH_SAMPLES 64
#define RAYMARCH_MAP_FNC(POS) raymarchMap(POS)
#define RAYMARCH_MAP_DISTANCE w
#define RAYMARCH_MAP_MATERIAL rgb

#define LIGHT_POSITION make_float3(0.0f, 10.0f, -50.0f)
#define LIGHT_COLOR make_float3(0.5f, 0.5f, 0.5f)
#define RAYMARCH_AMBIENT make_float3(1.0f, 1.0f, 1.0f)
#define RAYMARCH_BACKGROUND ( make_float3(0.7f, 0.9f, 1.0f) + ray.y * 0.8f )
// #define RAYMARCH_MAP_MATERIAL_TYPE float3

#include "lygia/lighting/camera.cuh"
#include "lygia/lighting/raymarch/cast.cuh"
#include "lygia/lighting/raymarch/normal.cuh"
#include "lygia/lighting/raymarch/ao.cuh"
#include "lygia/lighting/raymarch/softShadow.cuh"

__device__ float checkBoard(float2 _uv, float2 _scale) {
    _uv = floor(fract(_uv * _scale) * 2.0);
    return min(1.0, _uv.x + _uv.y) - (_uv.x * _uv.y);
}

__device__ float4 raymarchMap(const float3& pos) {
    float4 res = make_float4(1.0f);

    float check = checkBoard( make_float2(pos.x, pos.z), make_float2(1.0f));
    res = opUnion( res, make_float4( make_float3( 0.5f + check * 0.5f), planeSDF(pos) ) );
    res = opUnion( res, make_float4( 1.0f, 1.0f, 1.0f, sphereSDF( pos - make_float3( 0.0f, 0.60f, 0.0f), 0.5f ) ) );
        
    return res;
}

__device__ float3 raymarchMaterial(const float3& ray, const float3& position, const float3& normal, const float3& color) {
    float3  env = RAYMARCH_AMBIENT;

    if ( sum(color) <= 0.0f ) 
        return RAYMARCH_BACKGROUND;

    float3 ref = reflect( ray, normal );
    float occ = raymarchAO( position, normal );

    #if defined(LIGHT_DIRECTION)
    float3  lig = normalize( LIGHT_DIRECTION );
    #else
    float3  lig = normalize( LIGHT_POSITION - position);
    #endif
    
    float3 hal = normalize( lig-ray );
    float amb = saturate( 0.5f + 0.5f * normal.y );
    float dif = saturate( dot( normal, lig ) );
    float bac = saturate( dot( normal, normalize( make_float3(-lig.x, 0.0f,-lig.z))) ) * saturate( 1.0f - position.y );
    float dom = smoothstep( -0.1f, 0.1f, ref.y );
    float fre = pow( saturate(1.0+dot(normal,ray) ), 2.0f );
    
    dif *= raymarchSoftShadow( position, lig, 0.02f, 2.5f );
    dom *= raymarchSoftShadow( position, ref, 0.02f, 2.5f );

    float3 light = make_float3(0.0f);
    light += 1.30f * dif * LIGHT_COLOR;
    light += 0.40f * amb * occ * env;
    light += 0.50f * dom * occ * env;
    light += 0.50f * bac * occ * 0.25f;
    light += 0.25f * fre * occ;

    return color * light;
}


__global__ void render(int _width, int _height, float *_pixels, Camera _cam) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= _width || y >= _height)
        return;

    float2 resolution = make_float2(_width, _height);
    float2 pixel = 1.0f / resolution;
    float2 st = make_float2(x, y) * pixel;
    st = ratio(st, resolution);

    float2 uv = st * 2.0 - 1.0;
    float3 ray_origin = _cam.pos;
    float3 ray_direction = normalize(_cam.side * uv.x + _cam.up * uv.y + _cam.dir * _cam.invhalffov);

    float4 res = raymarchCast(ray_origin, ray_direction);
    float t = res.w;

    float3 pos = ray_origin + t * ray_direction;
    float3 nor = raymarchNormal( pos );

    float3 albedo = make_float3( res.x, res.y, res.z );
    float3 color = raymarchMaterial(ray_direction, pos, nor, albedo);
    // float3 color = nor;

    // return color
    _pixels[x * 4 + 4 * y * _width + 0] = color.x;
    _pixels[x * 4 + 4 * y * _width + 1] = color.y;
    _pixels[x * 4 + 4 * y * _width + 2] = color.z;
    _pixels[x * 4 + 4 * y * _width + 3] = 1.0f;
}

int main(int argc, char **argv) {
    int width = 1920;
    int height = 1080;

    // Create a buffer of pixels to be process on __device__
    float *d_pixels;
    hipMalloc(&d_pixels, 4 * width * height * sizeof(float));

    Camera cam;
	// cam.pos = make_float3(-1.0f, 1.5f, -1.0f);
	cam.pos = make_float3(0.0f, 1.0f, -1.6f);

	cam.dir = normalize( make_float3(0.0f, 0.0f, 0.6f) - cam.pos);
	cam.side = normalize(cross(cam.dir, make_float3(0.0f, 1.0f, 0.0f)));
	cam.up = normalize(cross(cam.side, cam.dir));
	float fov = 128.0f / 180.0f * float(M_PI);
	cam.invhalffov = 1.0f / std::tan(fov / 2.0f);

    // Hand the pixels to a kernel to be process on __device__ 
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(width / threads.x + 1, height / threads.y + 1);
    render<<<blocks, threads>>>(width, height, d_pixels, cam);

    // Copy pixels processed on __device__ into a HILMA Image on __host__
    hilma::Image image = hilma::Image(width, height, 4);
    hipMemcpy(&image[0], d_pixels, 4 * width * height * sizeof(float), hipMemcpyDeviceToHost);

    // Save image
    hilma::flip(image);
    hilma::savePng("image.png", image);

    // Free pixels on __device__
    hipFree(d_pixels);

    return 0;
}