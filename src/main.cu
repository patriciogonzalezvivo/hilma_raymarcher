#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <chrono>

#include <hip/hip_runtime.h>
#include <>
// #include "hiprand/hiprand_kernel.h"
#define BLOCK_SIZE 8

#include "hilma/types/Image.h"
#include "hilma/ops/convert_image.h"
#include "hilma/io/png.h"

#include "lygia/math/sum.cuh"
#include "lygia/math/cross.cuh"
#include "lygia/math/floor.cuh"
#include "lygia/math/fract.cuh"
#include "lygia/math/reflect.cuh"
#include "lygia/math/normalize.cuh"
#include "lygia/math/operations.cuh"
#include "lygia/math/smoothstep.cuh"
#include "lygia/space/ratio.cuh"
#include "lygia/sdf/sphereSDF.cuh"
#include "lygia/sdf/planeSDF.cuh"
#include "lygia/sdf/opUnion.cuh"
#include "lygia/color/palette/hue.cuh"
#include "lygia/lighting/camera.cuh"

#define RAYMARCH_MAP_TYPE float4
#define RAYMARCH_SAMPLES 64
#define RAYMARCH_MAP_FNC(POS) raymarchMap(POS)
#define RAYMARCH_MAP_DISTANCE w
#define RAYMARCH_MAP_MATERIAL rgb

#define LIGHT_POSITION make_float3(0.0f, 10.0f, -50.0f)
#define LIGHT_COLOR make_float3(0.5f, 0.5f, 0.5f)
#define RAYMARCH_AMBIENT make_float3(1.0f, 1.0f, 1.0f)
#define RAYMARCH_BACKGROUND ( make_float3(0.7f, 0.9f, 1.0f) + ray.y * 0.8f )
// #define RAYMARCH_MAP_MATERIAL_TYPE float3

__device__ float checkBoard(float2 _uv, float2 _scale) {
    _uv = floor(fract(_uv * _scale) * 2.0);
    return min(1.0, _uv.x + _uv.y) - (_uv.x * _uv.y);
}

__device__ float4 raymarchMap(const float3& pos ) {
    float4 res = make_float4(1.0f);

    float check = checkBoard( make_float2(pos.x, pos.z), make_float2(1.0f));
    res = opUnion( res, make_float4( make_float3( 0.5f + check * 0.5f), planeSDF(pos) ) );
    res = opUnion( res, make_float4( 1.0f, 1.0f, 1.0f, sphereSDF(    pos - make_float3( 0.0f, 0.60f, 0.0f), 0.5f ) ) );
        
    return res;
}

__device__ float4 raymarchCast(const float3& ro, const float3& rd) {
    float tmin = 1.0;
    float tmax = 20.0;
    
    float t = tmin;
    float4 m = make_float4(-1.0);
    for ( int i = 0; i < RAYMARCH_SAMPLES; i++ ) {
        float precis = 0.00001*t;
        RAYMARCH_MAP_TYPE res = RAYMARCH_MAP_FNC( ro + rd * t );
        if ( res.w < precis || t > tmax ) 
            break;
        t += res.w;
        m = res;
    }

    #if defined(RAYMARCH_BACKGROUND) || defined(RAYMARCH_FLOOR)
    if ( t > tmax ) 
        m = make_float4(-1.0);
    #endif

    m.w = t;
    return m;
}

__device__ float3 raymarchNormal(const float3& pos, float e) {
   const float2 offset = make_float2(1.0f, -1.0f);
   float3 offset_xyy = make_float3(offset.x, offset.y, offset.y);
   float3 offset_yyx = make_float3(offset.y, offset.y, offset.x);
   float3 offset_yxy = make_float3(offset.y, offset.x, offset.y);
   float3 offset_xxx = make_float3(offset.x, offset.x, offset.x);
   return normalize( offset_xyy * RAYMARCH_MAP_FNC( pos + offset_xyy * e ).RAYMARCH_MAP_DISTANCE +
                     offset_yyx * RAYMARCH_MAP_FNC( pos + offset_yyx * e ).RAYMARCH_MAP_DISTANCE +
                     offset_yxy * RAYMARCH_MAP_FNC( pos + offset_yxy * e ).RAYMARCH_MAP_DISTANCE +
                     offset_xxx * RAYMARCH_MAP_FNC( pos + offset_xxx * e ).RAYMARCH_MAP_DISTANCE );
}

__device__ float3 raymarchMaterial(float3 ray, float3 position, float3 normal, float3 color) {
    float3  env = RAYMARCH_AMBIENT;

    if ( sum(color) <= 0.0f ) 
        return RAYMARCH_BACKGROUND;

    float3 ref = reflect( ray, normal );
    float occ = 1.0f;//raymarchAO( position, normal );

    #if defined(LIGHT_DIRECTION)
    float3  lig = normalize( LIGHT_DIRECTION );
    #else
    float3  lig = normalize( LIGHT_POSITION - position);
    #endif
    
    float3 hal = normalize( lig-ray );
    float amb = saturate( 0.5f + 0.5f * normal.y );
    float dif = saturate( dot( normal, lig ) );
    float bac = saturate( dot( normal, normalize( make_float3(-lig.x, 0.0f,-lig.z))) ) * saturate( 1.0f - position.y );
    float dom = smoothstep( -0.1f, 0.1f, ref.y );
    float fre = pow( saturate(1.0+dot(normal,ray) ), 2.0f );
    
    // dif *= raymarchSoftShadow( position, lig, 0.02, 2.5 );
    // dom *= raymarchSoftShadow( position, ref, 0.02, 2.5 );

    float3 light = make_float3(0.0f);
    light += 1.30f * dif * LIGHT_COLOR;
    light += 0.40f * amb * occ * env;
    light += 0.50f * dom * occ * env;
    light += 0.50f * bac * occ * 0.25f;
    light += 0.25f * fre * occ;

    return color * light;
}

__device__ float3 raymarchNormal(const float3& pos) {
    return raymarchNormal(pos, 0.5773f * 0.0005f);
}

__global__ void render(int _width, int _height, float *_pixels, Camera _cam) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= _width || y >= _height)
        return;

    float2 resolution = make_float2(_width, _height);
    float2 pixel = 1.0f / resolution;
    float2 st = make_float2(x, y) * pixel;
    st = ratio(st, resolution);

    float2 uv = st * 2.0 - 1.0;
    float3 ray_origin = _cam.pos;
    float3 ray_direction = normalize(_cam.side * uv.x + _cam.up * uv.y + _cam.dir * _cam.invhalffov);

    float4 res = raymarchCast(ray_origin, ray_direction);
    float t = res.w;

    float3 pos = ray_origin + t * ray_direction;
    float3 nor = raymarchNormal( pos );

    float3 albedo = make_float3( res.x, res.y, res.z );
    // float3 color = raymarchMaterial(ray_direction, pos, nor, albedo);
    float3 color = nor;

    // return color
    _pixels[x * 4 + 4 * y * _width + 0] = color.x;
    _pixels[x * 4 + 4 * y * _width + 1] = color.y;
    _pixels[x * 4 + 4 * y * _width + 2] = color.z;
    _pixels[x * 4 + 4 * y * _width + 3] = 1.0f;
}

int main(int argc, char **argv) {
    int width = 1920;
    int height = 1080;

    // Create a buffer of pixels to be process on __device__
    float *d_pixels;
    hipMalloc(&d_pixels, 4 * width * height * sizeof(float));

    Camera cam;
	// cam.pos = make_float3(-1.0f, 1.5f, -1.0f);
	cam.pos = make_float3(0.0f, 1.0f, -0.6f);

	cam.dir = normalize( make_float3(0.0f, 0.0f, 0.6f) - cam.pos);
	cam.side = normalize(cross(cam.dir, make_float3(0.0f, 1.0f, 0.0f)));
	cam.up = normalize(cross(cam.side, cam.dir));
	float fov = 128.0f / 180.0f * float(M_PI);
	cam.invhalffov = 1.0f / std::tan(fov / 2.0f);

    // Hand the pixels to a kernel to be process on __device__ 
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(width / threads.x + 1, height / threads.y + 1);
    render<<<blocks, threads>>>(width, height, d_pixels, cam);

    // Copy pixels processed on __device__ into a HILMA Image on __host__
    hilma::Image image = hilma::Image(width, height, 4);
    hipMemcpy(&image[0], d_pixels, 4 * width * height * sizeof(float), hipMemcpyDeviceToHost);

    // Save image
    hilma::flip(image);
    hilma::savePng("image.png", image);

    // Free pixels on __device__
    hipFree(d_pixels);

    return 0;
}